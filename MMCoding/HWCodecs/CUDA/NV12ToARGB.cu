#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <stdint.h>

inline static __device__ float4 make_float4(uchar4 v)
{
    return make_float4(v.x, v.y, v.z, v.w);
}

inline static __device__ float2 make_float2(uchar2 v)
{
    return make_float2(v.x, v.y);
}

inline static __device__ uint8_t Clamp(float v)
{
    return max(min(__float2int_rn(v), 255), 0);
}

inline static __device__ uchar4 make_uchar4(float r, float g, float b)
{
    return make_uchar4(Clamp(r), Clamp(g), Clamp(b), 0xFF);
}

__device__ const float Y_OFFSET = -16.0f;
__device__ const float UV_OFFSET = -128.0f;
__device__ const float Y_SCALE = 1.1644f;
__device__ const float V_SCALE_R = 1.5960f;
__device__ const float V_SCALE_G = -0.8130f;
__device__ const float U_SCALE_G = -0.3918f;
__device__ const float U_SCALE_B = 2.0172f;

static __global__ void Nv12ToArgbKernel(const uint8_t* src, int srcPitch, hipSurfaceObject_t dst, int dstWidth, int dstHeight)
{
    int x = (blockIdx.x * blockDim.x + threadIdx.x) << 2;
    int y = (blockIdx.y * blockDim.y + threadIdx.y) << 1;
    if (x < dstWidth && y < dstHeight)
    {
        int surfaceX = x << 2;

        float4 nv = make_float4(*(uchar4*)(src + (dstHeight + (y >> 1)) * srcPitch + x)) + make_float4(UV_OFFSET);
        float4 yOffset = make_float4(Y_OFFSET);
        float4 u = make_float4(nv.x, nv.x, nv.z, nv.z);
        float4 v = make_float4(nv.y, nv.y, nv.w, nv.w);

        for (int row = 0; row < 2; ++row)
        {
            float4 luma = (make_float4(*(uchar4*)(src + (y + row) * srcPitch + x)) + yOffset) * Y_SCALE;

            float4 r = luma + v * V_SCALE_R;
            float4 g = luma + u * U_SCALE_G + v * V_SCALE_G;
            float4 b = luma + u * U_SCALE_B;

            surf2Dwrite(make_uchar4(r.x, g.x, b.x), dst, surfaceX,      y + row, hipBoundaryModeZero);
            surf2Dwrite(make_uchar4(r.y, g.y, b.y), dst, surfaceX + 4,  y + row, hipBoundaryModeZero);
            surf2Dwrite(make_uchar4(r.z, g.z, b.z), dst, surfaceX + 8,  y + row, hipBoundaryModeZero);
            surf2Dwrite(make_uchar4(r.w, g.w, b.w), dst, surfaceX + 12, y + row, hipBoundaryModeZero);
        }
    }
}

static __global__ void Nv12ToArgbScaleKernel(const uint8_t* src, int srcPitch, int srcHeight, hipSurfaceObject_t dst, int dstWidth, int dstHeight, int scale)
{
    int dstX = blockIdx.x * blockDim.x + threadIdx.x;
    int dstY = blockIdx.y * blockDim.y + threadIdx.y;
    if (dstX < dstWidth && dstY < dstHeight)
    {
        float y = ((*(src + (dstY * srcPitch + dstX) * scale)) + Y_OFFSET) * Y_SCALE;
        float2 nv = make_float2(*(uchar2*)(src + (srcHeight + ((dstY * scale) >> 1)) * srcPitch + (((dstX * scale) >> 1) << 1))) + make_float2(UV_OFFSET);
        float u = nv.x;
        float v = nv.y;

        float r = y + v * V_SCALE_R;
        float g = y + u * U_SCALE_G + v * V_SCALE_G;
        float b = y + u * U_SCALE_B;
        surf2Dwrite(make_uchar4(r, g, b), dst, dstX << 2, dstY, hipBoundaryModeZero);
    }
}

const int BLOCK_SIZE = 32;

dim3 DefaultBlock()
{
    return dim3(BLOCK_SIZE, BLOCK_SIZE);
}

dim3 GetGrid(int countX, int countY)
{
    return dim3((countX + BLOCK_SIZE - 1) / BLOCK_SIZE, (countY + BLOCK_SIZE - 1) / BLOCK_SIZE);
}

extern "C" void Nv12ToArgb(hipDeviceptr_t src, int srcWidth, int srcHeight, int srcPitch, hipArray_t dstArray)
{
    HIP_ARRAY_DESCRIPTOR dst = {};
    hipArrayGetDescriptor(&dst, dstArray);

    HIP_RESOURCE_DESC resourceDescriptor = {};
    resourceDescriptor.resType = HIPresourcetype::HIP_RESOURCE_TYPE_ARRAY;
    resourceDescriptor.res.array.hArray = dstArray;

    hipSurfaceObject_t surface = {};
    cuSurfObjectCreate(&surface, &resourceDescriptor);

    int scale = srcWidth / (int)dst.Width;
    if (scale == 1)
    {
        dim3 grid = GetGrid((int)dst.Width / 4, (int)dst.Height / 2);
        Nv12ToArgbKernel<<<grid, DefaultBlock()>>>((const uint8_t*)src, srcPitch, surface, (int)dst.Width, (int)dst.Height);
    }
    else
    {
        dim3 grid = GetGrid((int)dst.Width, (int)dst.Height);
        Nv12ToArgbScaleKernel<<<grid, DefaultBlock()>>>((const uint8_t*)src, srcPitch, srcHeight, surface, (int)dst.Width, (int)dst.Height, scale);
    }

    cuSurfObjectDestroy(surface);
}
